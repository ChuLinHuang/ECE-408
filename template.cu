#include "hip/hip_runtime.h"
#include <wb.h>

__global__ void vecAdd(float* in1, float* in2, float* out, int len) {

    int i;
    for (i = 0; i <= len; i = i + 1){ 
    	out[i] = in1[i] + in2[i];
    }
    //@@ Insert code to implement vector addition here
}

int main(int argc, char** argv) {
    wbArg_t args;
    int inputLength;
    float* hostInput1;
    float* hostInput2;
    float* hostOutput;
    float* deviceInput1;
    float* deviceInput2;
    float* deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 =
        (float*)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 =
        (float*)wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float*)malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

    wbTime_start(GPU, "Allocating GPU memory.");
    hipMalloc((void**)&deviceInput1, inputLength * sizeof(float));
    hipMalloc((void**)&deviceInput2, inputLength * sizeof(float));
    hipMalloc((void**)&deviceOutput, inputLength * sizeof(float));
    //@@ Allocate GPU memory here

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float), hipMemcpyHostToDevice);
    //@@ Copy memory to the GPU here

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    dim3 dimgrid(256, 1, 1);
    dim3 dimblock (256, 1, 1);
    //@@ Initialize the grid and block dimensions here

    wbTime_start(Compute, "Performing CUDA computation");
    vecAdd <<<dimgrid, dimblock >>> (deviceInput1, deviceInput2, deviceOutput, inputLength);
    //@@ Launch the GPU Kernel here

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float), hipMemcpyDeviceToHost);
    //@@ Copy the GPU memory back to the CPU here

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);
    //@@ Free the GPU memory here

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

